#include "hip/hip_runtime.h"
#include <stdio.h>
#include "wrapper.h"
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>

//int *hA, *hB;  // host input data
//int *hC;  // host output data

__global__ void vecAddition(int *A,int *B,int *C,int N)
{
   int id = blockIdx.x * blockDim.x + threadIdx.x;
   C[id] = A[id] + B[id]; 
}

int demo_kernel()
{
   // host input data
   int *hA, *hB;
   // host output data
   int *hC; 

   // Vector size
   int n=10000000;
   // Vector size in bytes
   int nBytes = n*sizeof(int);
  
   int thread_block, num_blocks; 

   // Allocate memory for host input data
   hA = (int *)malloc(nBytes);
   hB = (int *)malloc(nBytes);

   // Allocate memory for host output data
   hC = (int *)malloc(nBytes);

   // Initialize device input and output data
   int *dA,*dB,*dC;

   // Number of threads per block
   thread_block=512;

   // Number of blocks
   num_blocks = n/thread_block;

   // Vector initialization
   for(int i=0;i<n;i++){
      hA[i]=i;
      hB[i]=i;
   }

   // Allocate memory on GPU
   hipMalloc((void **)&dA, n*sizeof(int));
   hipMalloc((void **)&dB, n*sizeof(int));
   hipMalloc((void **)&dC, n*sizeof(int));

   // Copy host input data from host to device
   hipMemcpy(dA, hA, n*sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(dB, hB, n*sizeof(int), hipMemcpyHostToDevice);

   // Kernel launch
   vecAddition<<<num_blocks,thread_block>>>(dA,dB,dC,n);

   // Synchronize
   hipDeviceSynchronize()
   
   // Error checking
   hipError_t error = hipGetLastError();
   if(error != hipSuccess)
   {
    // print the CUDA error message and exit
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
   }

   // Copy output data from device to host
   hipMemcpy(hC, dC, n*sizeof(int), hipMemcpyDeviceToHost);

   // Free device memory
   hipFree(dA);
   hipFree(dB);
   hipFree(dC);

   // Free host memory
   free(hA);
   free(hB);
   free(hC);

   return 0;
}

